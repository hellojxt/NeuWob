#include "hip/hip_runtime.h"
#include "scene.h"
#include "nlohmann/json_fwd.hpp"
#include <nlohmann/json.hpp>
#include <fstream>
#define TINYOBJLOADER_IMPLEMENTATION
#include "tiny_obj_loader.h"
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <thrust/remove.h>
NWOB_NAMESPACE_BEGIN

SceneHost::SceneHost(const std::string config_json_file, int cut_idx)
{
    using json = nlohmann::json;
    json config;
    {
        std::ifstream config_file(config_json_file);
        if (!config_file)
        {
            std::cout << "Failed to load config file: " << config_json_file << std::endl;
            return;
        }
        config_file >> config;
        config_file.close();
    }
    const std::string input_obj_file = DATA_DIR + config["input_obj_file"].get<std::string>();
    tinyobj::ObjReader reader;
    if (!reader.ParseFromFile(input_obj_file))
    {
        std::cerr << "Failed to load " << input_obj_file << std::endl;
        exit(1);
    }
    if (!reader.Warning().empty())
    {
        std::cout << "WARN: " << reader.Warning() << std::endl;
    }

    const auto &attrib = reader.GetAttrib();
    const auto &shapes = reader.GetShapes();

    std::vector<float3> vertices(attrib.vertices.size() / 3);
    for (size_t v = 0; v < attrib.vertices.size() / 3; v++)
    {
        vertices[v] = make_float3(attrib.vertices[3 * v + 0], attrib.vertices[3 * v + 1], attrib.vertices[3 * v + 2]);
    }
    int triangle_num = 0;
    for (auto &shape : shapes)
    {
        triangle_num += shape.mesh.num_face_vertices.size();
    }

    std::vector<int3> triangles(triangle_num);

    for (auto &shape : shapes)
    {
        size_t index_offset = 0;
        for (size_t f = 0; f < shape.mesh.num_face_vertices.size(); f++)
        {
            int vertice_num_per_face = shape.mesh.num_face_vertices[f];
            assert(vertice_num_per_face == 3);
            tinyobj::index_t idx0 = shape.mesh.indices[f * 3 + 0];
            tinyobj::index_t idx1 = shape.mesh.indices[f * 3 + 1];
            tinyobj::index_t idx2 = shape.mesh.indices[f * 3 + 2];
            triangles[f + index_offset] = make_int3(idx0.vertex_index, idx1.vertex_index, idx2.vertex_index);
        }
        index_offset += shape.mesh.num_face_vertices.size();
    }

    std::cout << "Vertices number: " << vertices.size() << "\n";
    std::cout << "Triangles number: " << triangles.size() << "\n";
    // Loaded success
    std::cout << "OBJ file:" << input_obj_file << " loaded!"
              << "\n";

    std::vector<Element> elements;
    if (cut_idx >= 1)
        elements.resize(cut_idx);
    else
        elements.resize(triangles.size());

    for (int i = 0; i < elements.size(); i++)
    {
        elements[i].v0 = vertices[triangles[i].x];
        elements[i].v1 = vertices[triangles[i].y];
        elements[i].v2 = vertices[triangles[i].z];
        elements[i].n = normalize(cross(elements[i].v1 - elements[i].v0, elements[i].v2 - elements[i].v0));
        elements[i].indices = make_int3(triangles[i].x, triangles[i].y, triangles[i].z);
    }
    elements_device.resize_and_copy_from_host(elements);
    std::vector<float> area_cdf_host(elements.size());
    total_area = 0.f;
    for (int i = 0; i < elements.size(); i++)
    {
        float area = elements[i].area();
        total_area += area;
        area_cdf_host[i] = total_area;
    }
    area_cdf.resize_and_copy_from_host(area_cdf_host);
    size_t boundary_point_num = config["boundary_point_num"].get<int>();
    boundary_points_device.resize(boundary_point_num);
    printf("Number of boundary points: %ld\n", boundary_point_num);
    sample_boundary_points();

    auto grid_min_point = config["grid_min_point"].get<std::vector<float>>();
    auto grid_max_point = config["grid_max_point"].get<std::vector<float>>();
    grid.min_pos = make_float3(grid_min_point[0], grid_min_point[1], grid_min_point[2]);
    grid.max_pos = make_float3(grid_max_point[0], grid_max_point[1], grid_max_point[2]);
    auto grid_resolution = config["grid_resolution"].get<int>();
    grid.size = make_int3(grid_resolution, grid_resolution, grid_resolution);
    grid.cell_length = ((grid.max_pos - grid.min_pos) / make_float3(grid.size)).x;
    construct_neighbor_list();
    printf("Grid size: %d %d %d\n", grid.size.x, grid.size.y, grid.size.z);
}

void SceneHost::save_boundary_points(const std::string filename) const
{
    std::vector<BoundaryPoint> boundary_points_host;
    boundary_points_host.resize(boundary_points_device.size());
    std::ofstream ofs(filename);
    boundary_points_device.copy_to_host(boundary_points_host);
    for (auto &bp : boundary_points_host)
    {
        ofs << bp.pos.x << " " << bp.pos.y << " " << bp.pos.z << "\n";
    }
}

void SceneHost::sample_boundary_points()
{
    size_t boundary_point_num = boundary_points_device.size();
    GPUMemory<unsigned long long> seeds(boundary_point_num);
    seeds.copy_from_host(get_random_seeds(boundary_point_num));
    parallel_for(boundary_point_num, [scene = device(), seeds = seeds.device_ptr()] __device__(int i) {
        auto seed = seeds[i];
        randomState rand_state;
        hiprand_init(seed, 0, 0, &rand_state);
        scene.boundary_points[i] = scene.sample_boundary_point(&rand_state);
    });
}

void SceneHost::construct_neighbor_list()
{
    GPUMemory<NeighborList> self_list(grid.get_cell_num());
    self_list.memset(0);
    parallel_for(boundary_points_device.size(), [grid = grid, bps = boundary_points_device.device_ptr(),
                                                 self_list = self_list.device_ptr()] __device__(int i) {
        auto bp = bps[i];
        auto grid_index = grid.get_flat_index(bp.pos);
        self_list[grid_index].atomic_append(i);
    });
    GPUMemory<int> non_empty(grid.get_cell_num());
    parallel_for(grid.get_cell_num(),
                 [self_list = self_list.device_ptr(), non_empty = non_empty.device_ptr()] __device__(int i) {
                     if (self_list[i].size() > 0)
                         non_empty[i] = i + 1;
                     else
                         non_empty[i] = 0;
                 });
    auto last_iter = thrust::remove(thrust::device, non_empty.begin(), non_empty.end(), 0);
    int non_empty_cell_num = last_iter - non_empty.begin();
    neighbor_list.resize(grid.get_cell_num());
    neighbor_list.memset(0);
    parallel_for(non_empty_cell_num, [non_empty = non_empty.device_ptr(), self_list = self_list.device_ptr(),
                                      neighbor_list = neighbor_list.device_ptr(), grid = grid] __device__(int i) {
        int flat_idx = non_empty[i] - 1;
        int3 cell_idx = grid.get_cell_index(flat_idx);
        for (int x = -1; x <= 1; x++)
        {
            for (int y = -1; y <= 1; y++)
            {
                for (int z = -1; z <= 1; z++)
                {
                    int3 neighbor_cell_idx = cell_idx + make_int3(x, y, z);
                    auto &src_lst = self_list[grid.get_flat_index(neighbor_cell_idx)];
                    for (int j = 0; j < src_lst.size(); j++)
                    {
                        neighbor_list[flat_idx].append(src_lst[j]);
                    }
                }
            }
        }
    });
}
NWOB_NAMESPACE_END
