#include "hip/hip_runtime.h"
#include "scene.h"
#include "nlohmann/json_fwd.hpp"
#include <nlohmann/json.hpp>
#include <fstream>
#define TINYOBJLOADER_IMPLEMENTATION
#include "tiny_obj_loader.h"
#include <vector>

NWOB_NAMESPACE_BEGIN

SceneHost::SceneHost(const std::string config_json_file, int cut_idx)
{
    using json = nlohmann::json;
    json config;
    {
        std::ifstream config_file(config_json_file);
        if (!config_file)
        {
            std::cout << "Failed to load config file: " << config_json_file << std::endl;
            return;
        }
        config_file >> config;
        config_file.close();
    }
    const std::string input_obj_file = DATA_DIR + config["input_obj_file"].get<std::string>();
    tinyobj::ObjReader reader;
    if (!reader.ParseFromFile(input_obj_file))
    {
        std::cerr << "Failed to load " << input_obj_file << std::endl;
        exit(1);
    }
    if (!reader.Warning().empty())
    {
        std::cout << "WARN: " << reader.Warning() << std::endl;
    }

    const auto &attrib = reader.GetAttrib();
    const auto &shapes = reader.GetShapes();

    std::vector<float3> vertices(attrib.vertices.size() / 3);
    for (size_t v = 0; v < attrib.vertices.size() / 3; v++)
    {
        vertices[v] = make_float3(attrib.vertices[3 * v + 0], attrib.vertices[3 * v + 1], attrib.vertices[3 * v + 2]);
    }
    int triangle_num = 0;
    for (auto &shape : shapes)
    {
        triangle_num += shape.mesh.num_face_vertices.size();
    }

    std::vector<int3> triangles(triangle_num);

    for (auto &shape : shapes)
    {
        size_t index_offset = 0;
        for (size_t f = 0; f < shape.mesh.num_face_vertices.size(); f++)
        {
            int vertice_num_per_face = shape.mesh.num_face_vertices[f];
            assert(vertice_num_per_face == 3);
            tinyobj::index_t idx0 = shape.mesh.indices[f * 3 + 0];
            tinyobj::index_t idx1 = shape.mesh.indices[f * 3 + 1];
            tinyobj::index_t idx2 = shape.mesh.indices[f * 3 + 2];
            triangles[f + index_offset] = make_int3(idx0.vertex_index, idx1.vertex_index, idx2.vertex_index);
        }
        index_offset += shape.mesh.num_face_vertices.size();
    }

    std::cout << "Vertices number: " << vertices.size() << "\n";
    std::cout << "Triangles number: " << triangles.size() << "\n";
    // Loaded success
    std::cout << "OBJ file:" << input_obj_file << " loaded!"
              << "\n";

    elements.clear();
    for (int i = 0; i < triangles.size(); i++)
    {
        Element e;
        e.v0 = vertices[triangles[i].x];
        e.v1 = vertices[triangles[i].y];
        e.v2 = vertices[triangles[i].z];
        e.n = normalize(cross(e.v1 - e.v0, e.v2 - e.v0));
        e.indices = make_int3(triangles[i].x, triangles[i].y, triangles[i].z);
        elements.push_back(e);
    }
    if (cut_idx >= 0)
    {
        elements.erase(elements.begin() + cut_idx, elements.end());
    }
}

NWOB_NAMESPACE_END
