#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include "imgui.h"
#define IMGUI_DEFINE_MATH_OPERATORS
#include <stdio.h>
#include "backends/imgui_impl_glfw.h"
#include "backends/imgui_impl_opengl3.h"
#include "imgui_internal.h"
#include "common.h"
#include "ui.h"
#include "gpu_memory.h"
#include "png.h"
#include <stdlib.h>
#include <cstring>

NWOB_NAMESPACE_BEGIN
static void glfw_error_callback(int error, const char *description)
{
    fprintf(stderr, "Glfw Error %d: %s\n", error, description);
}

void MemoryVisualizer::visualize(GPUMatrix<uchar4> *data)
{
    gpu_matrix = data;
    auto width = data->width();
    auto height = data->height();
    std::cout << "width: " << width << " height: " << height << std::endl;
    glfwSetErrorCallback(glfw_error_callback);
    if (!glfwInit())
        return;
    std::cout << "GLFW initialized" << std::endl;
    const char *glsl_version = "#version 130";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
    // glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);  // 3.2+ only
    // glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);            // 3.0+ only
    GLFWwindow *window = glfwCreateWindow(1280, 720, "Dear ImGui Visualizer", NULL, NULL);
    if (window == NULL)
        return;

    glfwMakeContextCurrent(window);

    glfwSwapInterval(1);  // Enable vsync
    if (glewInit() != GLEW_OK)
        exit(EXIT_FAILURE);

    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO &io = ImGui::GetIO();
    (void)io;
    io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;  // Enable Keyboard Controls

    // Setup Dear ImGui style
    // ImGui::StyleColorsDark();
    ImGui::StyleColorsClassic();

    // Setup Platform/Renderer bindings
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init(glsl_version);
    ImVec4 clear_color = ImVec4(0.45f, 0.55f, 0.60f, 1.00f);
    io.IniFilename = IMGUI_CONFIG_FILE;
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &image);
    glBindTexture(GL_TEXTURE_2D, image);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
    glBindTexture(GL_TEXTURE_2D, 0);
    CUDA_CHECK_THROW(
        hipGraphicsGLRegisterImage(&CudaResource, image, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
    CUDA_CHECK_THROW(hipGraphicsMapResources(1, &CudaResource, 0));
    CUDA_CHECK_THROW(hipGraphicsSubResourceGetMappedArray(&array, CudaResource, 0, 0));

    while (!glfwWindowShouldClose(window))
    {
        glfwPollEvents();

        // Start the Dear ImGui frame
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();
        // ImGui::Text("FPS: %.4f", ImGui::GetIO().Framerate);
        ImGui::Begin("Memory Visualizer");
        CUDA_CHECK_THROW(hipMemcpy2DToArray(array, 0, 0, gpu_matrix->data(), width * sizeof(uchar4),
                                             height * sizeof(uchar4), height, hipMemcpyDeviceToDevice));
        ImVec2 wsize = ImGui::GetWindowContentRegionMax() - ImGui::GetWindowContentRegionMin();
        ImVec2 img_size = ImVec2(wsize.x, wsize.y - ImGui::GetFrameHeightWithSpacing() * 2);
        if (img_size.x < img_size.y)
        {
            img_size.y = img_size.x * height / width;
        }
        else
        {
            img_size.x = img_size.y * width / height;
        }
        ImGui::SetCursorPos((wsize - img_size) * 0.5f);
        ImGui::Image((ImTextureID)(uintptr_t)image, img_size, ImVec2(0, 1), ImVec2(1, 0));
        ImGui::End();

        // Rendering
        ImGui::Render();
        int display_w, display_h;
        glfwGetFramebufferSize(window, &display_w, &display_h);
        glViewport(0, 0, display_w, display_h);
        glClearColor(clear_color.x * clear_color.w, clear_color.y * clear_color.w, clear_color.z * clear_color.w,
                     clear_color.w);
        glClear(GL_COLOR_BUFFER_BIT);
        // glUseProgram(0); // You may want this if using this code in an OpenGL 3+ context where shaders may be bound
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

        glfwSwapBuffers(window);
    }

    // Cleanup
    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();

    glfwDestroyWindow(window);
    glfwTerminate();
}

void MemoryVisualizer::write_to_png(const char *filename, GPUMatrix<uchar4> *data)
{
    auto width = data->width();
    auto height = data->height();
    std::vector<uchar4> data_host(width * height);
    data->memory.copy_to_host(data_host);
    FILE *fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8, PNG_COLOR_TYPE_RGBA, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);
    png_write_info(png_ptr, info_ptr);
    png_bytep row = (png_bytep)malloc(4 * width * sizeof(png_byte));
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            row[j * 4 + 0] = data_host[i * width + j].x;
            row[j * 4 + 1] = data_host[i * width + j].y;
            row[j * 4 + 2] = data_host[i * width + j].z;
            row[j * 4 + 3] = data_host[i * width + j].w;
        }
        png_write_row(png_ptr, row);
    }
    png_write_end(png_ptr, NULL);
    fclose(fp);
    free(row);
}

NWOB_NAMESPACE_END
