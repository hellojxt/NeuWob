#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include "imgui.h"
#define IMGUI_DEFINE_MATH_OPERATORS
#include <stdio.h>
#include "backends/imgui_impl_glfw.h"
#include "backends/imgui_impl_opengl3.h"
#include "imgui_internal.h"
#include "common.h"
#include "ui.h"
#include "gpu_memory.h"

NWOB_NAMESPACE_BEGIN
static void glfw_error_callback(int error, const char *description)
{
    fprintf(stderr, "Glfw Error %d: %s\n", error, description);
}

void MemoryVisualizer::visualize(GPUMatrix<uchar4> *data)
{
    gpu_matrix = data;
    auto width = data->width();
    auto height = data->height();
    std::cout << "width: " << width << " height: " << height << std::endl;
    glfwSetErrorCallback(glfw_error_callback);
    if (!glfwInit())
        return;
    std::cout << "GLFW initialized" << std::endl;
    const char *glsl_version = "#version 130";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
    // glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);  // 3.2+ only
    // glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);            // 3.0+ only
    GLFWwindow *window = glfwCreateWindow(1280, 720, "Dear ImGui Visualizer", NULL, NULL);
    if (window == NULL)
        return;

    glfwMakeContextCurrent(window);

    glfwSwapInterval(1);  // Enable vsync
    if (glewInit() != GLEW_OK)
        exit(EXIT_FAILURE);

    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO &io = ImGui::GetIO();
    (void)io;
    io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;  // Enable Keyboard Controls

    // Setup Dear ImGui style
    // ImGui::StyleColorsDark();
    ImGui::StyleColorsClassic();

    // Setup Platform/Renderer bindings
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init(glsl_version);
    ImVec4 clear_color = ImVec4(0.45f, 0.55f, 0.60f, 1.00f);
    io.IniFilename = IMGUI_CONFIG_FILE;
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &image);
    glBindTexture(GL_TEXTURE_2D, image);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
    glBindTexture(GL_TEXTURE_2D, 0);
    CUDA_CHECK_THROW(
        hipGraphicsGLRegisterImage(&CudaResource, image, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
    CUDA_CHECK_THROW(hipGraphicsMapResources(1, &CudaResource, 0));
    CUDA_CHECK_THROW(hipGraphicsSubResourceGetMappedArray(&array, CudaResource, 0, 0));

    while (!glfwWindowShouldClose(window))
    {
        glfwPollEvents();

        // Start the Dear ImGui frame
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();
        // ImGui::Text("FPS: %.4f", ImGui::GetIO().Framerate);
        ImGui::Begin("Memory Visualizer");
        CUDA_CHECK_THROW(hipMemcpy2DToArray(array, 0, 0, gpu_matrix->data(), width * sizeof(uchar4),
                                             height * sizeof(uchar4), height, hipMemcpyDeviceToDevice));
        ImVec2 wsize = ImGui::GetWindowContentRegionMax() - ImGui::GetWindowContentRegionMin();
        ImVec2 img_size = ImVec2(wsize.x, wsize.y - ImGui::GetFrameHeightWithSpacing() * 2);
        if (img_size.x < img_size.y)
        {
            img_size.y = img_size.x * height / width;
        }
        else
        {
            img_size.x = img_size.y * width / height;
        }
        ImGui::SetCursorPos((wsize - img_size) * 0.5f);
        ImGui::Image((ImTextureID)(uintptr_t)image, img_size, ImVec2(0, 1), ImVec2(1, 0));
        ImGui::End();

        // Rendering
        ImGui::Render();
        int display_w, display_h;
        glfwGetFramebufferSize(window, &display_w, &display_h);
        glViewport(0, 0, display_w, display_h);
        glClearColor(clear_color.x * clear_color.w, clear_color.y * clear_color.w, clear_color.z * clear_color.w,
                     clear_color.w);
        glClear(GL_COLOR_BUFFER_BIT);
        // glUseProgram(0); // You may want this if using this code in an OpenGL 3+ context where shaders may be bound
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

        glfwSwapBuffers(window);
    }

    // Cleanup
    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();

    glfwDestroyWindow(window);
    glfwTerminate();
}

NWOB_NAMESPACE_END
