#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "gpu_memory.h"
#include "ui.h"
using namespace nwob;

int main()
{
    GPUMatrix<uchar4> image(512, 512);
    image.memory.memset(125);
    parallel_for(image.size(), [out = image.device_ptr()] __device__(int i) {
        int y = i / out.stride();
        int x = i % out.stride();
        out[y][x] = make_uchar4(y / 2, 0, 0, 255);
    });
    MemoryVisualizer().visualize(&image);
}