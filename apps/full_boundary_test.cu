#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "gpu_memory.h"
#include "ui.h"
#include "estimator.h"
#include "scene.h"
#include "external/integrand.h"

using namespace nwob;
#define MOLTIPOLR_M 0
#define COLOR_SCALE 10
int main(int argc, char *argv[])
{

    std::string input_json_file;
    if (argc != 2)
    {
        input_json_file = "../data/test.json";
    }
    else
        input_json_file = argv[1];
    SceneHost scene_host(input_json_file);
    float k = 10;
    float3 x0 = make_float3(0, 0, 0);
    scene_host.set_neumann([&](float3 p, float3 n) { return Green_func_deriv<HELMHOLTZ>(x0, p, n, k); });
    GPUMemory<unsigned long long> seeds(1);
    seeds.copy_from_host(get_random_seeds(1));
    printf("seeds copied\n");
    parallel_for(1, [seeds = seeds.device_ptr(), x0, k, scene = scene_host.device()] __device__(int i) {
        int spp1 = 5, spp2 = 100000;
        int path_length = 5;
        complex sum = 0;
        auto seed = seeds[i];
        auto &bvh = scene.bvh;
        BoundaryPoint bp, pre, next, src;
        randomState rand_state;
        hiprand_init(seed, 0, 0, &rand_state);
        float inv_pdf;
        auto dirichlet_func = [x0] __device__(float3 p, float k) { return Green_func<HELMHOLTZ>(x0, p, k); };
        for (int i = 0; i < spp1; i++)
        {
            int trg_id = hiprand(&rand_state) % bvh.num_objects;
            complex b_dirichlet = 0;
            pre.pos = bvh.objects[trg_id].center();
            for (int j = 0; j < spp2; j++)
            {
                thrust::tie(next, inv_pdf) = scene.uniform_sample(&rand_state, pre);
                b_dirichlet +=
                    inv_pdf * 2 *
                    (Green_func_deriv<HELMHOLTZ>(pre.pos, next.pos, next.normal, k) * dirichlet_func(next.pos, k) -
                     Green_func<HELMHOLTZ>(pre.pos, next.pos, k) * next.neumann);
            }
            b_dirichlet /= spp2;
            printf("b_dirichlet: %e + %ei\n", b_dirichlet.real(), b_dirichlet.imag());

            complex bem_dirichlet = 0;
            complex gt_dirichlet = dirichlet_func(bvh.objects[trg_id].center(), k);
            printf("gt_dirichlet: %e + %ei\n", gt_dirichlet.real(), gt_dirichlet.imag());

            for (int j = 0; j < bvh.num_objects; j++)
            {
                bem_dirichlet += face2FaceIntegrand(bvh.objects[j], bvh.objects[trg_id], k, nwob::DOUBLE_LAYER) *
                                     dirichlet_func(bvh.objects[j].center(), k) -
                                 face2FaceIntegrand(bvh.objects[j], bvh.objects[trg_id], k, nwob::SINGLE_LAYER) *
                                     bvh.objects[j].center_neumann();
            }
            bem_dirichlet /= (0.5f * bvh.objects[trg_id].area());
            printf("bem_dirichlet: %e + %ei\n", bem_dirichlet.real(), bem_dirichlet.imag());
        }
    });
}