#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "gpu_memory.h"
#include "ui.h"
#include "estimator.h"
#include "scene.h"
#include "multipole.h"
#include "external/integrand.h"

using namespace nwob;
#define MOLTIPOLR_M 0
#define COLOR_SCALE 10
int main(int argc, char *argv[])
{

    std::string input_json_file;
    if (argc != 2)
    {
        input_json_file = "../data/test.json";
    }
    else
        input_json_file = argv[1];
    SceneHost scene_host(input_json_file, 1);
    float3 x0 = {0.0f, 0.0f, 0.0f};
    float k = 10;
    scene_host.set_neumann([&](float3 p, float3 n) { return 1.0f; });
    int res = 1;
    GPUMatrix<uchar4> image(res, res);
    float3 grid_min_pos = {0, -2, -2};
    float width = 4;
    float3 dx = {0, width / res, 0};
    float3 dy = {0, 0, width / res};
    GPUMemory<unsigned long long> seeds(res * res);
    seeds.copy_from_host(get_random_seeds(res * res));
    printf("seeds copied\n");
    parallel_for(res * res, [seeds = seeds.device_ptr(), res, x0, grid_min_pos, dx, dy, k, scene = scene_host.device(),
                             out = image.device_ptr()] __device__(int i) {
        int x = i % res;
        int y = i / res;
        // printf("x: %d, y: %d\n", x, y);
        float3 p = grid_min_pos + x * dx + y * dy;
        // printf("p: %f, %f, %f\n", p.x, p.y, p.z);
        int spp = 100000;
        int path_length = 5;
        complex sum = 0;
        auto seed = seeds[i];
        randomState rand_state;
        hiprand_init(seed, 0, 0, &rand_state);
        auto tri = scene.bvh.objects[0];
        complex dirichlet = face2FaceIntegrand(tri, tri, k, nwob::SINGLE_LAYER) * tri.neumann(0.5f, 0.5f) /
                            (face2FaceIntegrand(tri, tri, k, nwob::DOUBLE_LAYER) - 0.5f * tri.area());

        printf("dirichlet: %e + %ei\n", dirichlet.real(), dirichlet.imag());

        BoundaryPoint bp, pre, next;
        float inv_pdf;
        for (int i = 0; i < spp; i++)
        {
            thrust::tie(bp, inv_pdf) = scene.uniform_sample(&rand_state, bp);
            pre = bp;
            thrust::tie(next, inv_pdf) = scene.uniform_sample(&rand_state, pre);
            sum += -inv_pdf * 2 * Green_func<HELMHOLTZ>(pre.pos, next.pos, k) * next.neumann;
            thrust::tie(next, inv_pdf) = scene.uniform_sample(&rand_state, pre);
            sum += inv_pdf * 2 * Green_func_deriv<HELMHOLTZ>(pre.pos, next.pos, next.normal, k) * dirichlet;
        }
        sum /= spp;
        printf("sum: %e + %ei\n", sum.real(), sum.imag());
        float v = sum.real() * COLOR_SCALE + 0.5;
        out[x][y] = get_viridis_color(v);
    });
}