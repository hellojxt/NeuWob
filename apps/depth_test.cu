#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "gpu_memory.h"
#include "ui.h"
#include "estimator.h"
#include "scene.h"
#include "external/integrand.h"

using namespace nwob;
#define COLOR_SCALE 4

constexpr int equation_type = HELMHOLTZ;
auto G0 = Green_func<equation_type>;
auto G1 = Green_func_deriv<equation_type>;

class Reservoir
{
    public:
        uint y, M;
        float w_sum;
        HOST_DEVICE Reservoir() : y(0), M(0), w_sum(0) {}
        inline HOST_DEVICE void update(uint x, float w, float rd)
        {
            w_sum += w;
            M++;
            if (rd < w / w_sum)
                y = x;
        }
};

int main()
{
    std::string input_json_file = "../data/test.json";
    SceneHost scene_host(input_json_file);
    float3 x0 = {0.0f, 0.0f, 0.0f};
    float k = 10;
    scene_host.set_neumann([&](float3 p, float3 n) { return G1(x0, p, n, k); });
    int res = 256;
    GPUMatrix<uchar4> image(res, res);
    float3 grid_min_pos = {0, -2, -2};
    float width = 4;
    float3 dx = {0, width / res, 0};
    float3 dy = {0, 0, width / res};
    GPUMemory<unsigned long long> seeds(res * res);
    seeds.copy_from_host(get_random_seeds(res * res));

    std::vector<NeighborList> neighbor_list_host;
    std::vector<BoundaryPoint> boundary_points_host;
    neighbor_list_host.resize(scene_host.neighbor_list.size());
    scene_host.neighbor_list.copy_to_host(neighbor_list_host.data());
    boundary_points_host.resize(scene_host.boundary_points_device.size());
    scene_host.boundary_points_device.copy_to_host(boundary_points_host.data());

    int idx = 0;
    for (auto bp : boundary_points_host)
    {
        idx++;
        auto pos = bp.pos;
        auto &lst = neighbor_list_host[scene_host.grid.get_flat_index(pos)];
        std::ofstream ofs(std::string("../output/neighbor_list_") + std::to_string(idx) + ".txt");
        for (int i = 0; i < lst.size(); i++)
        {
            auto &p = boundary_points_host[lst[i]];
            ofs << p.pos.x << " " << p.pos.y << " " << p.pos.z << std::endl;
        }
        ofs << pos.x << " " << pos.y << " " << pos.z << std::endl;
    }

    // parallel_for(res * res, [seeds = seeds.device_ptr(), res, x0, grid_min_pos, dx, dy, k, scene =
    // scene_host.device(),
    //                          out = image.device_ptr()] __device__(int i) {
    //     int x = i % res;
    //     int y = i / res;
    //     float3 p = grid_min_pos + x * dx + y * dy;
    //     int spp = 100000;
    //     complex sum = 0;
    //     auto seed = seeds[i];
    //     randomState rand_state;
    //     hiprand_init(seed, 0, 0, &rand_state);
    //     for (int i = 0; i < spp; i++)
    //     {
    //         float inv_pdf;
    //         BoundaryPoint src, bp;
    //         thrust::tie(bp, inv_pdf) = scene.uniform_sample(&rand_state, src);
    //         sum += -inv_pdf * Green_func<HELMHOLTZ>(p, bp.pos, k) * bp.neumann;
    //         complex weight = inv_pdf * Green_func_deriv<HELMHOLTZ>(p, bp.pos, bp.normal, k);

    //         while (true)
    //         {
    //             float ksi = hiprand_uniform(&rand_state);
    //             float P_RR = 0.0001f;
    //             if (ksi > P_RR)
    //                 break;
    //             // if (i == path_depth - 1)
    //             //     weight *= 0.5f;
    //             //
    //             // if (i == path_depth - 1)
    //             // {
    //             //     sum += weight * Green_func<HELMHOLTZ>(x0, bp.pos, k);
    //             //     break;
    //             // }
    //             BoundaryPoint dst;
    //             thrust::tie(dst, inv_pdf) = scene.uniform_sample(&rand_state, bp);
    //             inv_pdf *= 2 / P_RR;
    //             sum += weight * (-inv_pdf * Green_func<HELMHOLTZ>(bp.pos, dst.pos, k) * dst.neumann);
    //             weight *= inv_pdf * Green_func_deriv<HELMHOLTZ>(bp.pos, dst.pos, dst.normal, k);
    //             bp = dst;
    //         }
    //     }
    //     sum /= spp;
    //     if (x == 0 && y == 0)
    //         printf("sum: %e\n", sum.real());
    //     float v = sum.real() * COLOR_SCALE + 0.5;

    //     out[x][y] = get_viridis_color(v);
    // });
    // MemoryVisualizer().write_to_png("../output/wob_real.png", &image);

    // parallel_for(res * res, [res, x0, grid_min_pos, dx, dy, k, scene = scene_host.device(),
    //                          out = image.device_ptr()] __device__(int i) {
    //     int x = i % res;
    //     int y = i / res;
    //     float3 p = grid_min_pos + x * dx + y * dy;
    //     complex sum = 0;
    //     for (int i = 0; i < scene.bvh.num_objects; i++)
    //     {
    //         auto &obj = scene.bvh.objects[i];
    //         float3 c = (obj.v0 + obj.v1 + obj.v2) / 3;
    //         complex dirichlet = Green_func<HELMHOLTZ>(x0, c, k).real();
    //         complex neumann = Green_func_deriv<HELMHOLTZ>(x0, c, obj.n, k).real();
    //         sum += face2PointIntegrand(obj, p, k, nwob::DOUBLE_LAYER) * dirichlet -
    //                face2PointIntegrand(obj, p, k, nwob::SINGLE_LAYER) * neumann;
    //     }
    //     if (x == 0 && y == 0)
    //         printf("sum: %e\n", sum.real());
    //     float v = sum.real() * COLOR_SCALE + 0.5;
    //     out[x][y] = get_viridis_color(v);
    // });

    // MemoryVisualizer().write_to_png("../output/bem_real.png", &image);

    // parallel_for(res * res, [x0, grid_min_pos, dx, dy, k, res, out = image.device_ptr()] __device__(int i) {
    //     int x = i % res;
    //     int y = i / res;
    //     float3 p = grid_min_pos + x * dx + y * dy;
    //     float v = Green_func<HELMHOLTZ>(x0, p, k).real() * COLOR_SCALE + 0.5;
    //     out[x][y] = get_viridis_color(v);
    // });
    // MemoryVisualizer().write_to_png("../output/gt_real.png", &image);
}