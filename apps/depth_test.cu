#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "gpu_memory.h"
#include "ui.h"
#include "estimator.h"
#include "scene.h"
#include "external/integrand.h"

using namespace nwob;
#define COLOR_SCALE 4

class Reservoir
{
    public:
        uint y, M;
        float w_sum;
        float W;
        HOST_DEVICE Reservoir() : y(0), M(0), w_sum(0), W(0) {}
        inline HOST_DEVICE void update(uint xi, float wi, float rd)
        {
            w_sum += wi;
            M++;
            if (rd <= wi / w_sum)
                y = xi;
        }
};

class ReservoirPair
{
    public:
        Reservoir dirichlet, neumann;
};

void set_ground_truth(SceneHost &scene_host, float3 x0, float k)
{
    parallel_for(scene_host.boundary_points_device.size(), [x0, k, scene = scene_host.device()] __device__(int i) {
        auto &bp = scene.boundary_points[i];
        bp.dirichlet = Green_func<HELMHOLTZ>(x0, bp.pos, k);
    });
}

int main()
{
    std::string input_json_file = "../data/test.json";
    SceneHost scene_host(input_json_file);
    float3 x0 = {0.0f, 0.0f, 0.0f};
    float k = 10;
    parallel_for(scene_host.boundary_points_device.size(), [x0, k, scene = scene_host.device()] __device__(int i) {
        auto &bp = scene.boundary_points[i];
        bp.neumann = Green_func_deriv<HELMHOLTZ>(x0, bp.pos, bp.normal, k);
    });
    int res = 256;
    GPUMatrix<uchar4> image(res, res);
    float3 grid_min_pos = {0, -2, -2};
    float width = 4;
    float3 dx = {0, width / res, 0};
    float3 dy = {0, 0, width / res};

    int max_iter = 1000;
    int RIS_M = 32;
    int path_depth = 1;
    // std::cin >> path_depth;
    int points_num = scene_host.boundary_points_device.size();
    GPUMemory<ReservoirPair> reservoirs(points_num);
    reservoirs.memset(0);
    GPUMemory<unsigned long long> seeds(points_num);
    seeds.copy_from_host(get_random_seeds(points_num));
    GPUMemory<randomState> rand_states(points_num);
    parallel_for(points_num, [seeds = seeds.device_ptr(), rand_states = rand_states.device_ptr()] __device__(int i) {
        hiprand_init(seeds[i], 0, 0, &rand_states[i]);
    });

    for (int iter_idx = 0; iter_idx < max_iter; iter_idx++)
    {
        // RIS
        // parallel_for(points_num, [RIS_M, k, scene = scene_host.device(), seeds = seeds.device_ptr(),
        //                           reservoirs = reservoirs.device_ptr()] __device__(int i) {
        //     auto &bp = scene.boundary_points[i];
        //     randomState rand_state;
        //     hiprand_init(seeds[i], 0, 0, &rand_state);
        //     ReservoirPair r;
        //     for (int j = 0; j < RIS_M; j++)
        //     {
        //         int xi = hiprand_uniform(&rand_state) * scene.num_boundary_points;
        //         auto src = scene.boundary_points[xi];
        //         float rd = hiprand_uniform(&rand_state);
        //         // float p_hat0 = max(abs(Green_func_deriv<HELMHOLTZ>(bp.pos, src.pos, src.normal, k).real()),
        //         1e-3f); float p_hat0 = 1.0f;
        //         // float p_hat1 = max(abs(Green_func<HELMHOLTZ>(bp.pos, src.pos, k).real()), 1e-3f);
        //         float p_hat1 = 1.0f;
        //         float p_sample = 1.0f / scene.total_area;
        //         r.dirichlet.update(xi, p_hat0 / p_sample, rd);
        //         r.neumann.update(xi, p_hat1 / p_sample, rd);
        //     }
        //     auto src = scene.boundary_points[r.dirichlet.y];
        //     // float p_hat = max(abs(Green_func_deriv<HELMHOLTZ>(bp.pos, src.pos, src.normal, k).real()), 1e-3f);
        //     float p_hat = 1.0f;
        //     r.dirichlet.W = 1.0f / p_hat * (r.dirichlet.w_sum / r.dirichlet.M);
        //     src = scene.boundary_points[r.neumann.y];
        //     // p_hat = max(abs(Green_func<HELMHOLTZ>(bp.pos, src.pos, k).real()), 1e-3f);
        //     p_hat = 1.0f;
        //     r.neumann.W = 1.0f / p_hat * (r.neumann.w_sum / r.neumann.M);
        //     reservoirs[i] = r;
        // });

        // Spatial reuse

        // Compute boundary points value
        parallel_for(points_num, [iter_idx, max_iter, x0, k, path_depth, scene = scene_host.device(),
                                  rand_states = rand_states.device_ptr(),
                                  reservoirs = reservoirs.device_ptr()] __device__(int i) {
            int trg_idx = i;
            complex sum = 0;
            complex weight = 1.0f;
            auto &trg = scene.boundary_points[trg_idx];
            // auto &src_dirichlet = scene.boundary_points[reservoirs[trg_idx].dirichlet.y];
            // auto &src_neumann = scene.boundary_points[reservoirs[trg_idx].neumann.y];
            int xi = scene.sample_points_index(&rand_states[i]);
            auto &src_dirichlet = scene.boundary_points[xi];
            xi = scene.sample_points_index(&rand_states[i]);
            auto &src_neumann = scene.boundary_points[xi];
            reservoirs[i].dirichlet.W = scene.total_area;
            reservoirs[i].neumann.W = scene.total_area;
            sum = 2 *
                  (reservoirs[i].dirichlet.W *
                       Green_func_deriv<HELMHOLTZ>(trg.pos, src_dirichlet.pos, src_dirichlet.normal, k) *
                       Green_func<HELMHOLTZ>(src_dirichlet.pos, x0, k) -
                   reservoirs[i].neumann.W * Green_func<HELMHOLTZ>(trg.pos, src_neumann.pos, k) * src_neumann.neumann);
            // for (int j = 0; j < path_depth; j++)
            // {
            //     if (j == path_depth - 1)
            //         weight *= 0.5f;  // path truncation in WOB
            //     auto &trg = scene.boundary_points[trg_idx];
            //     auto &src = scene.boundary_points[reservoirs[trg_idx].y];
            //     sum += -weight * 2 * reservoirs[i].W * Green_func<HELMHOLTZ>(trg.pos, src.pos, k) *
            //     src.neumann; weight *= 2 * reservoirs[i].W * Green_func_deriv<HELMHOLTZ>(trg.pos, src.pos,
            //     src.normal, k); trg_idx = reservoirs[trg_idx].y;
            // }
            scene.boundary_points[i].dirichlet += sum;
            if (iter_idx == max_iter - 1)
                scene.boundary_points[i].dirichlet /= max_iter;
        });
    }

    // Compute domain points value
    // set_ground_truth(scene_host, x0, k);
    parallel_for(res * res, [res, x0, grid_min_pos, dx, dy, k, scene = scene_host.device(),
                             out = image.device_ptr()] __device__(int i) {
        int x = i % res;
        int y = i / res;
        float3 p = grid_min_pos + x * dx + y * dy;
        complex sum = 0;
        int sample_num = scene.num_boundary_points;
        for (int i = 0; i < sample_num; i++)
        {
            auto bp = scene.boundary_points[i];
            sum += scene.total_area * (Green_func_deriv<HELMHOLTZ>(p, bp.pos, bp.normal, k) * bp.dirichlet -
                                       0 * Green_func<HELMHOLTZ>(p, bp.pos, k) * bp.neumann);
        }
        sum /= sample_num;
        if (x == 0 && y == 0)
            printf("%f\n", sum.real());
        float v = sum.real() * COLOR_SCALE + 0.5;
        out[x][y] = get_viridis_color(v);
    });
    MemoryVisualizer().write_to_png("../output/wob_real.png", &image);

    parallel_for(res * res, [res, x0, grid_min_pos, dx, dy, k, scene = scene_host.device(),
                             out = image.device_ptr()] __device__(int i) {
        int x = i % res;
        int y = i / res;
        float3 p = grid_min_pos + x * dx + y * dy;
        complex sum = 0;
        for (int i = 0; i < scene.num_elements; i++)
        {
            auto &obj = scene.elements[i];
            float3 c = (obj.v0 + obj.v1 + obj.v2) / 3;
            complex dirichlet = Green_func<HELMHOLTZ>(x0, c, k).real();
            complex neumann = Green_func_deriv<HELMHOLTZ>(x0, c, obj.n, k).real();
            sum += face2PointIntegrand(obj, p, k, nwob::DOUBLE_LAYER) * dirichlet -
                   0 * face2PointIntegrand(obj, p, k, nwob::SINGLE_LAYER) * neumann;
        }
        if (x == 0 && y == 0)
            printf("%f\n", sum.real());
        float v = sum.real() * COLOR_SCALE + 0.5;
        out[x][y] = get_viridis_color(v);
    });

    MemoryVisualizer().write_to_png("../output/bem_real.png", &image);

    parallel_for(res * res, [x0, grid_min_pos, dx, dy, k, res, out = image.device_ptr()] __device__(int i) {
        int x = i % res;
        int y = i / res;
        float3 p = grid_min_pos + x * dx + y * dy;
        float v = Green_func<HELMHOLTZ>(x0, p, k).real() * COLOR_SCALE + 0.5;
        out[x][y] = get_viridis_color(v);
    });
    MemoryVisualizer().write_to_png("../output/gt_real.png", &image);
}