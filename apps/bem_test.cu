#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "gpu_memory.h"
#include "ui.h"
#include "estimator.h"
#include "scene.h"
#include "external/integrand.h"

using namespace nwob;
#define MOLTIPOLR_M 0
#define COLOR_SCALE 10

int main(int argc, char *argv[])
{

    std::string input_json_file;
    if (argc != 2)
    {
        input_json_file = "../data/test.json";
    }
    else
        input_json_file = argv[1];

    SceneHost scene_host(input_json_file);
    float k = 10;
    float3 x0 = {0, 0, 0};
    scene_host.set_neumann([&](float3 p, float3 n) { return Green_func_deriv<HELMHOLTZ>(x0, p, n, k); });

    complex bem_dirichlet = 0;
    int trg_id = 2;
    auto &e_lst = scene_host.elements;
    auto &trg_e = e_lst[trg_id];
    printf("trg_e: (%f, %f, %f) (%f, %f, %f) (%f, %f, %f)\n", trg_e.v0.x, trg_e.v0.y, trg_e.v0.z, trg_e.v1.x,
           trg_e.v1.y, trg_e.v1.z, trg_e.v2.x, trg_e.v2.y, trg_e.v2.z);
    for (int i = 0; i < e_lst.size(); i++)
    {
        auto &src_e = e_lst[i];
        complex double_layer = face2FaceIntegrand(src_e, trg_e, k, nwob::DOUBLE_LAYER);
        complex dirichlet = Green_func<HELMHOLTZ>(src_e.center(), x0, k);
        complex single_layer = face2FaceIntegrand(src_e, trg_e, k, nwob::SINGLE_LAYER);
        complex neumann = src_e.center_neumann();
        complex contrib = double_layer * dirichlet - single_layer * neumann;
        bem_dirichlet += contrib;
    }
    printf("bem_dirichlet: %e + %ei\n", bem_dirichlet.real(), bem_dirichlet.imag());
    complex gt_dirichlet = Green_func<HELMHOLTZ>(trg_e.center(), x0, k);
    gt_dirichlet *= (0.5f * trg_e.area());
    printf("gt_dirichlet: %e + %ei\n", gt_dirichlet.real(), gt_dirichlet.imag());
}
