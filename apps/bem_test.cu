#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "gpu_memory.h"
#include "ui.h"
#include "estimator.h"
#include "scene.h"
#include "external/integrand.h"

using namespace nwob;
#define MOLTIPOLR_M 0
#define COLOR_SCALE 10

int main(int argc, char *argv[])
{

    std::string input_json_file;
    if (argc != 2)
    {
        input_json_file = "../data/test.json";
    }
    else
        input_json_file = argv[1];

    SceneHost scene_host(input_json_file);
    float k = 10;
    float3 x0 = {0, 0, 0};
    parallel_for(scene_host.elements_device.size(), [x0, k, scene = scene_host.device()] __device__(int i) {
        auto &e = scene.elements[i];
        e.N0 = Green_func_deriv<HELMHOLTZ>(x0, e.v0, e.n, k);
        e.N1 = Green_func_deriv<HELMHOLTZ>(x0, e.v1, e.n, k);
        e.N2 = Green_func_deriv<HELMHOLTZ>(x0, e.v2, e.n, k);
    });

    complex bem_dirichlet = 0;
    int trg_id = 2;
    std::vector<Element> e_lst(scene_host.elements_device.size());
    scene_host.elements_device.copy_to_host(e_lst.data());
    auto &trg_e = e_lst[trg_id];
    printf("trg_e: (%f, %f, %f) (%f, %f, %f) (%f, %f, %f)\n", trg_e.v0.x, trg_e.v0.y, trg_e.v0.z, trg_e.v1.x,
           trg_e.v1.y, trg_e.v1.z, trg_e.v2.x, trg_e.v2.y, trg_e.v2.z);
    for (int i = 0; i < e_lst.size(); i++)
    {
        auto &src_e = e_lst[i];
        complex double_layer = face2FaceIntegrand(src_e, trg_e, k, nwob::DOUBLE_LAYER);
        complex dirichlet = Green_func<HELMHOLTZ>(src_e.center(), x0, k);
        complex single_layer = face2FaceIntegrand(src_e, trg_e, k, nwob::SINGLE_LAYER);
        complex neumann = src_e.center_neumann();
        complex contrib = double_layer * dirichlet - single_layer * neumann;
        bem_dirichlet += contrib;
    }
    printf("bem_dirichlet: %e + %ei\n", bem_dirichlet.real(), bem_dirichlet.imag());
    complex gt_dirichlet = Green_func<HELMHOLTZ>(trg_e.center(), x0, k);
    gt_dirichlet *= (0.5f * trg_e.area());
    printf("gt_dirichlet: %e + %ei\n", gt_dirichlet.real(), gt_dirichlet.imag());
}
