#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "gpu_memory.h"
#include "ui.h"
#include "estimator.h"
#include "scene.h"
#include "external/integrand.h"

using namespace nwob;
#define MOLTIPOLR_M 0
#define COLOR_SCALE 10

int main(int argc, char *argv[])
{

    std::string input_json_file;
    if (argc != 2)
    {
        input_json_file = "../data/test.json";
    }
    else
        input_json_file = argv[1];
    SceneHost scene_host(input_json_file);
    float3 x0 = {0.0f, 0.0f, 0.0f};
    float k = 10;
    scene_host.set_neumann([&](float3 p, float3 n) { return Green_func_deriv<POSSION>(x0, p, n, k); });
    int res = 256;
    GPUMatrix<uchar4> image(res, res);
    float3 grid_min_pos = {0, -2, -2};
    float width = 4;
    float3 dx = {0, width / res, 0};
    float3 dy = {0, 0, width / res};
    GPUMemory<unsigned long long> seeds(res * res);
    seeds.copy_from_host(get_random_seeds(res * res));
    printf("seeds copied\n");
    int path_depth;
    std::cin >> path_depth;
    parallel_for(res * res, [path_depth, seeds = seeds.device_ptr(), res, x0, grid_min_pos, dx, dy, k,
                             scene = scene_host.device(), out = image.device_ptr()] __device__(int i) {
        int x = i % res;
        int y = i / res;
        float3 p = grid_min_pos + x * dx + y * dy;
        int spp = 100000;
        complex sum = 0;
        auto seed = seeds[i];
        randomState rand_state;
        hiprand_init(seed, 0, 0, &rand_state);
        for (int i = 0; i < spp; i++)
        {
            float inv_pdf;
            BoundaryPoint src, bp;
            thrust::tie(bp, inv_pdf) = scene.uniform_sample(&rand_state, src);
            sum += -inv_pdf * Green_func<POSSION>(p, bp.pos, k) * bp.neumann;
            complex weight = inv_pdf * Green_func_deriv<POSSION>(p, bp.pos, bp.normal, k);

            for (int i = 0; i < path_depth; i++)
            {
                if (i == path_depth - 1)
                    weight *= 0.5f;
                // {
                //     sum += weight * Green_func<POSSION>(x0, bp.pos, k);
                //     break;
                // }

                BoundaryPoint dst;
                thrust::tie(dst, inv_pdf) = scene.uniform_sample(&rand_state, bp);
                inv_pdf *= 2;
                sum += weight * (-inv_pdf * Green_func<POSSION>(bp.pos, dst.pos, k) * dst.neumann);
                weight *= inv_pdf * Green_func_deriv<POSSION>(bp.pos, dst.pos, dst.normal, k);
                bp = dst;
            }
        }
        sum /= spp;
        if (x == 0 && y == 0)
            printf("sum: %e\n", sum.real());
        float v = sum.real() * COLOR_SCALE;

        out[x][y] = get_viridis_color(v);
    });
    MemoryVisualizer().write_to_png("../output/wob_real.png", &image);

    parallel_for(res * res, [x0, grid_min_pos, dx, dy, k, res, out = image.device_ptr()] __device__(int i) {
        int x = i % res;
        int y = i / res;
        float3 p = grid_min_pos + x * dx + y * dy;
        float v = Green_func<POSSION>(x0, p, k).real() * COLOR_SCALE;
        out[x][y] = get_viridis_color(v);
    });
    MemoryVisualizer().write_to_png("../output/gt_real.png", &image);
}