#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "gpu_memory.h"
#include "ui.h"
#include "estimator.h"
#include "scene.h"
#include "external/integrand.h"

using namespace nwob;
#define COLOR_SCALE 4
int main()
{
    std::string input_json_file = "../data/test.json";
    SceneHost scene_host(input_json_file);
    float3 x0 = {0.0f, 0.0f, 0.0f};
    float k = 10;
    parallel_for(scene_host.elements_device.size(), [x0, k, scene = scene_host.device()] __device__(int i) {
        auto &e = scene.elements[i];
        e.N0 = Green_func_deriv<HELMHOLTZ>(x0, e.v0, e.n, k);
        e.N1 = Green_func_deriv<HELMHOLTZ>(x0, e.v1, e.n, k);
        e.N2 = Green_func_deriv<HELMHOLTZ>(x0, e.v2, e.n, k);
    });
    scene_host.save_boundary_points("../output/boundary_points.txt");
    int res = 256;
    GPUMatrix<uchar4> image(res, res);
    float3 grid_min_pos = {0, -2, -2};
    float width = 4;
    float3 dx = {0, width / res, 0};
    float3 dy = {0, 0, width / res};
    GPUMemory<unsigned long long> seeds(res * res);
    seeds.copy_from_host(get_random_seeds(res * res));

    std::vector<NeighborList> neighbor_list_host;
    std::vector<BoundaryPoint> boundary_points_host;
    neighbor_list_host.resize(scene_host.neighbor_list.size());
    scene_host.neighbor_list.copy_to_host(neighbor_list_host.data());
    boundary_points_host.resize(scene_host.boundary_points_device.size());
    scene_host.boundary_points_device.copy_to_host(boundary_points_host.data());

    int idx = 0;
    for (auto bp : boundary_points_host)
    {
        idx++;
        auto pos = bp.pos;
        auto &lst = neighbor_list_host[scene_host.grid.get_flat_index(pos)];
        std::ofstream ofs(std::string("../output/neighbor_list_") + std::to_string(idx) + ".txt");
        for (int i = 0; i < lst.size(); i++)
        {
            auto &p = boundary_points_host[lst[i]];
            ofs << p.pos.x << " " << p.pos.y << " " << p.pos.z << std::endl;
        }
        if (lst.size() == 0)
            printf("no neighbor for %d\n", idx);
        ofs << pos.x << " " << pos.y << " " << pos.z << std::endl;
    }
}