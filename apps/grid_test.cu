#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "gpu_memory.h"
#include "ui.h"
#include "estimator.h"
#include "scene.h"
#include "external/integrand.h"

using namespace nwob;
#define COLOR_SCALE 4

constexpr int equation_type = HELMHOLTZ;
auto G0 = Green_func<equation_type>;
auto G1 = Green_func_deriv<equation_type>;

int main()
{
    std::string input_json_file = "../data/test.json";
    SceneHost scene_host(input_json_file);
    float3 x0 = {0.0f, 0.0f, 0.0f};
    float k = 10;
    scene_host.set_neumann([&](float3 p, float3 n) { return G1(x0, p, n, k); });
    int res = 256;
    GPUMatrix<uchar4> image(res, res);
    float3 grid_min_pos = {0, -2, -2};
    float width = 4;
    float3 dx = {0, width / res, 0};
    float3 dy = {0, 0, width / res};
    GPUMemory<unsigned long long> seeds(res * res);
    seeds.copy_from_host(get_random_seeds(res * res));

    std::vector<NeighborList> neighbor_list_host;
    std::vector<BoundaryPoint> boundary_points_host;
    neighbor_list_host.resize(scene_host.neighbor_list.size());
    scene_host.neighbor_list.copy_to_host(neighbor_list_host.data());
    boundary_points_host.resize(scene_host.boundary_points_device.size());
    scene_host.boundary_points_device.copy_to_host(boundary_points_host.data());

    int idx = 0;
    for (auto bp : boundary_points_host)
    {
        idx++;
        auto pos = bp.pos;
        auto &lst = neighbor_list_host[scene_host.grid.get_flat_index(pos)];
        std::ofstream ofs(std::string("../output/neighbor_list_") + std::to_string(idx) + ".txt");
        for (int i = 0; i < lst.size(); i++)
        {
            auto &p = boundary_points_host[lst[i]];
            ofs << p.pos.x << " " << p.pos.y << " " << p.pos.z << std::endl;
        }
        if (lst.size() == 0)
            printf("no neighbor for %d\n", idx);
        ofs << pos.x << " " << pos.y << " " << pos.z << std::endl;
    }
}