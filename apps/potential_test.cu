#include "hip/hip_runtime.h"
#include <iostream>
#include "common.h"
#include "gpu_memory.h"
#include "ui.h"
#include "estimator.h"
#include "scene.h"
#include "multipole.h"
#include "external/integrand.h"

using namespace nwob;
#define MOLTIPOLR_M 1
int main(int argc, char *argv[])
{

    std::string input_json_file;
    if (argc != 2)
    {
        input_json_file = "../data/test.json";
    }
    else
        input_json_file = argv[1];
    SceneHost scene_host(input_json_file);
    float3 x0 = {0.0f, 0.0f, 0.0f};
    float k = 10;
    scene_host.set_neumann(
        [&](float3 p, float3 n) { return multipole_basis_deriv<MOLTIPOLR_M, 0>(x0, p, k, n).real(); });
    int res = 256;
    GPUMatrix<uchar4> image(res, res);
    float3 grid_min_pos = {0, -2, -2};
    float width = 4;
    float3 dx = {0, width / res, 0};
    float3 dy = {0, 0, width / res};
    GPUMemory<unsigned long long> seeds(res * res);
    seeds.copy_from_host(get_random_seeds(res * res));
    printf("seeds copied\n");
    parallel_for(res * res, [seeds = seeds.device_ptr(), res, x0, grid_min_pos, dx, dy, k, scene = scene_host.device(),
                             out = image.device_ptr()] __device__(int i) {
        int x = i % res;
        int y = i / res;
        // printf("x: %d, y: %d\n", x, y);
        float3 p = grid_min_pos + x * dx + y * dy;
        // printf("p: %f, %f, %f\n", p.x, p.y, p.z);
        int spp = 20000;
        int path_length = 10;
        complex sum = 0;
        auto seed = seeds[i];
        Estimator es(scene, seed, path_length);
        for (int i = 0; i < spp; i++)
        {
            sum += es.compute_domain_value(
                p, k, [x0] __device__(float3 p, float k) { return multipole_basis<MOLTIPOLR_M, 0>(x0, p, k).real(); });
        }
        sum /= spp;
        if (x == 0 && y == 0)
            printf("sum: %e\n", sum.real());
        float v = sum.real() * 4 + 0.5;

        out[x][y] = get_viridis_color(v);
    });
    MemoryVisualizer().write_to_png("../output/wob_real.png", &image);

    parallel_for(res * res, [res, x0, grid_min_pos, dx, dy, k, scene = scene_host.device(),
                             out = image.device_ptr()] __device__(int i) {
        int x = i % res;
        int y = i / res;
        float3 p = grid_min_pos + x * dx + y * dy;
        complex sum = 0;
        for (int i = 0; i < scene.bvh.num_objects; i++)
        {
            auto &obj = scene.bvh.objects[i];
            float3 c = (obj.v0 + obj.v1 + obj.v2) / 3;
            complex dirichlet = multipole_basis<MOLTIPOLR_M, 0>(x0, c, k).real();
            complex neumann = multipole_basis_deriv<MOLTIPOLR_M, 0>(x0, c, k, obj.n).real();
            sum += face2PointIntegrand(obj, p, k, nwob::DOUBLE_LAYER) * dirichlet -
                   face2PointIntegrand(obj, p, k, nwob::SINGLE_LAYER) * neumann;
            // printf("face2PointIntegrand(obj, p, k, nwob::DOUBLE_LAYER): %e, %e\n",
            //        face2PointIntegrand(obj, p, k, nwob::DOUBLE_LAYER).real(),
            //        face2PointIntegrand(obj, p, k, nwob::DOUBLE_LAYER).imag());
            // printf("dirichlet: %e\n", dirichlet.real());
        }
        if (x == 0 && y == 0)
            printf("sum: %e\n", sum.real());
        float v = sum.real() * 4 + 0.5;
        out[x][y] = get_viridis_color(v);
    });

    MemoryVisualizer().write_to_png("../output/bem_real.png", &image);

    parallel_for(res * res, [x0, grid_min_pos, dx, dy, k, res, out = image.device_ptr()] __device__(int i) {
        int x = i % res;
        int y = i / res;
        float3 p = grid_min_pos + x * dx + y * dy;
        float v = multipole_basis<MOLTIPOLR_M, 0>(x0, p, k).real() * 4 + 0.5;
        out[x][y] = get_viridis_color(v);
    });
    MemoryVisualizer().write_to_png("../output/gt_real.png", &image);
}